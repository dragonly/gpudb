#ifndef SCAN_IMPL_CU
#define SCAN_IMPL_CU

#include "common.h"
#include "gpuCudaLib.h"
// this is embarrassingly working by means of stick three .cu files together
#include "scan.cu"

void static scanImpl(int *d_input, int rLen, int *d_output, struct statistic *pp) {
  int len = 2;
  if (rLen < len) {
    int *input, *output;
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&input, len * sizeof(int)));
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&output, len * sizeof(int)));
    CUDA_SAFE_CALL_NO_SYNC(hipMemset(input, 0, len * sizeof(int)));
    CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(input, d_input, rLen * sizeof(int), hipMemcpyDeviceToDevice));
    preallocBlockSums(len);
    prescanArray(output, input, len, pp);
    deallocBlockSums();
    CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(d_output, output, rLen * sizeof(int), hipMemcpyDeviceToDevice));
    CUDA_SAFE_CALL_NO_SYNC(hipFree(input));
    CUDA_SAFE_CALL_NO_SYNC(hipFree(output));
    return;
  } else {
    preallocBlockSums(rLen);
    prescanArray(d_output, d_input, rLen, pp);
    deallocBlockSums();
  }
  //	preallocBlockSums(rLen);
  //	prescanArray(d_output, d_input, rLen, pp);
  //	deallocBlockSums();
}

#endif
