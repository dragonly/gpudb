#include "hip/hip_runtime.h"
/*
   Copyright (c) 2012-2013 The Ohio State University.

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

       http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License.
*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <time.h>
#include "common.h"
#include "gpuCudaLib.h"
#include "scanImpl.cu"

__device__ static unsigned int StringHash(const char *s) {
  unsigned int hash = 0;
  int c;
  while ((c = *s++)) {
    hash = ((hash << 5) + hash) ^ c;
  }
  return hash;
}

/*
 * Transform integer to string using one single gpu thread.
 */
__device__ static char *gpuItoa(int value, char *result, int base) {

  if (base < 2 || base > 36) {
    *result = '\0';
    return result;
  }

  char *ptr = result, *ptr1 = result, tmp_char;
  int tmp_value;

  do {
    tmp_value = value;
    value /= base;
    *ptr++ = "zyxwvutsrqponmlkjihgfedcba9876543210123456789abcdefghijklmnopqrstuvwxyz"[35 + (tmp_value - value * base)];
  } while (value);

  if (tmp_value < 0)
    *ptr++ = '-';

  *ptr-- = '\0';

  while (ptr1 < ptr) {
    tmp_char = *ptr;
    *ptr-- = *ptr1;
    *ptr1++ = tmp_char;
  }
  return result;
}

/*
 * string copy using one gpu thread.
 */

//__device__ static char *gpuStrcpy(char *dst, const char *src) {
//
//  char *orig = dst;
//  while (*src)
//    *dst++ = *src++;
//  *dst = '\0';
//
//  return orig;
//}

__device__ static char *gpuStrncat(char *dest, const char *src, size_t n) {
  int dest_len = 0;
  int i;

  char *tmp = dest;
  while (*tmp != '\0') {
    tmp++;
    dest_len++;
  }

  for (i = 0; i < n && src[i] != '\0'; i++)
    dest[dest_len + i] = src[i];
  dest[dest_len + i] = '\0';
  return dest;
}

__device__ static char *gpuStrcat(char *dest, const char *src) {
  char *tmp = dest;
  int dest_len = 0;
  int i;

  while (*tmp != '\0') {
    tmp++;
    dest_len++;
  }

  for (i = 0; src[i] != '\0'; i++) {
    dest[dest_len + i] = src[i];
  }

  dest[dest_len + i] = '\0';

  return dest;
}

/*
 * Combine the group by columns to build the group by keys.
 */

extern "C" __global__ void build_groupby_key(char **content, int gbColNum, int *gbIndex, int *gbType, int *gbSize,
                                             long tupleNum, int *key, int *num) {

  int stride = blockDim.x * gridDim.x;
  int offset = blockIdx.x * blockDim.x + threadIdx.x;

  for (long i = offset; i < tupleNum; i += stride) {
    char buf[128] = { 0 };
    for (int j = 0; j < gbColNum; j++) {
      char tbuf[32] = { 0 };
      int index = gbIndex[j];

      if (index == -1) {
        gpuItoa(1, tbuf, 10);
        gpuStrncat(buf, tbuf, 1);

      } else if (gbType[j] == STRING) {
        gpuStrncat(buf, content[index] + i * gbSize[j], gbSize[j]);

      } else if (gbType[j] == INT) {
        int key = ((int *)(content[index]))[i];
        gpuItoa(key, tbuf, 10);
        gpuStrcat(buf, tbuf);
      }
    }
    int hkey = StringHash(buf) % HSIZE;
    key[i] = hkey;
    num[hkey] = 1;
  }
}

/*
 * This is for testing only.
 */

extern "C" __global__ void build_groupby_key_soa(char **content, int gbColNum, int *gbIndex, int *gbType, int *gbSize,
                                                 long tupleNum, int *key, int *num) {

  int stride = blockDim.x * gridDim.x;
  int offset = blockIdx.x * blockDim.x + threadIdx.x;

  for (long i = offset; i < tupleNum; i += stride) {
    char buf[128] = { 0 };
    for (int j = 0; j < gbColNum; j++) {
      char tbuf[32] = { 0 };
      int index = gbIndex[j];

      if (index == -1) {
        gpuItoa(1, tbuf, 10);
        gpuStrncat(buf, tbuf, 1);

      } else if (gbType[j] == STRING) {
        for (int k = 0; k < gbSize[j]; k++) {
          long pos = k * tupleNum + i;
          buf[k] = content[index][pos];
        }
        gpuStrncat(buf, tbuf, gbSize[j]);

      } else if (gbType[j] == INT) {
        int key = ((int *)(content[index]))[i];
        gpuItoa(key, tbuf, 10);
        gpuStrcat(buf, tbuf);
      }
    }
    int hkey = StringHash(buf) % HSIZE;
    key[i] = hkey;
    num[hkey] = 1;
  }
}

/*
 * Count the number of groups
 */

extern "C" __global__ void count_group_num(int *num, int tupleNum, int *totalCount) {
  int stride = blockDim.x * gridDim.x;
  int offset = blockIdx.x * blockDim.x + threadIdx.x;
  int localCount = 0;

  for (int i = offset; i < tupleNum; i += stride) {
    if (num[i] == 1) {
      localCount++;
    }
  }

  atomicAdd(totalCount, localCount);
}

/*
 * Calculate the groupBy expression.
 */

__device__ static float calMathExp(char **content, struct mathExp *exp, int pos, int op) {
  float res;

  if (op == NOOP) {
    if (exp[0].opType == CONS)
      res = exp[0].opValue;
    else {
      int index = exp[0].opValue;
      res = ((int *)(content[index]))[pos];
    }

  } else if (op == PLUS) {
    res = calMathExp(content, &exp[0], pos, NOOP) + calMathExp(content, &exp[1], pos, NOOP);

  } else if (op == MINUS) {
    res = calMathExp(content, &exp[0], pos, NOOP) - calMathExp(content, &exp[1], pos, NOOP);

  } else if (op == MULTIPLY) {
    res = calMathExp(content, &exp[0], pos, NOOP) * calMathExp(content, &exp[1], pos, NOOP);

  } else if (op == DIVIDE) {
    res = calMathExp(content, &exp[0], pos, NOOP) / calMathExp(content, &exp[1], pos, NOOP);
  }

  return res;
}

/*
 * group by constant. Currently only support SUM function.
 */

extern "C" __global__ void agg_cal_cons(char **content, int colNum, int *funcArray, int *op, struct mathExp *exp,
                                        int *mathOffset, int *gbType, int *gbSize, long tupleNum, int *key, int *psum,
                                        char **result) {

  int stride = blockDim.x * gridDim.x;
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  float buf[32];
  for (int i = 0; i < 32; i++)
    buf[i] = 0;

  for (int i = index; i < tupleNum; i += stride) {
    for (int j = 0; j < colNum; j++) {
      int func = funcArray[j];
      int offset = mathOffset[j];
      if (func == SUM) {
        float tmpRes = calMathExp(content, &exp[offset], i, op[j]);
        buf[j] += tmpRes;
      }
    }
  }

  for (int i = 0; i < colNum; i++)
    atomicAdd(&((float *)result[i])[0], buf[i]);
}

/*
 * gropu by
 */

extern "C" __global__ void agg_cal(char **content, int colNum, int *funcArray, int *op, struct mathExp *exp,
                                   int *mathOffset, int *gbType, int *gbSize, long tupleNum, int *key, int *psum,
                                   char **result) {

  int stride = blockDim.x * gridDim.x;
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  for (int i = index; i < tupleNum; i += stride) {

    int hKey = key[i];
    int offset = psum[hKey];

    for (int j = 0; j < colNum; j++) {
      int func = funcArray[j];
      int mo = mathOffset[j];
      if (func == NOOP) {
        int type = exp[mo].opType;

        if (type == CONS) {
          int value = exp[mo].opValue;
          ((int *)result[j])[offset] = value;
        } else {
          int index = exp[mo].opValue;
          int attrSize = gbSize[j];
          if (attrSize == sizeof(int))
            ((int *)result[j])[offset] = ((int *)content[index])[i];
          else
            memcpy(result[j] + offset * attrSize, content[index] + i * attrSize, attrSize);
        }

      } else if (func == SUM) {
        float tmpRes = calMathExp(content, &exp[mo], i, op[j]);
        atomicAdd(&((float *)result[j])[offset], tmpRes);
      }
    }
  }
}

/*
 * groupBy: group by the data and calculate.
 *
 * Prerequisite:
 *  input data are not compressed
 *
 * Input:
 *  gb: the groupby node which contains the input data and groupby information
 *  pp: records the statistics such as kernel execution time
 *
 * Return:
 *  a new table node
 */

struct tableNode *groupBy(struct groupByNode *gb, struct statistic *pp) {
  struct timespec start, end;
  clock_gettime(CLOCK_REALTIME, &start);
  int *gpuGbIndex = NULL, gpuTupleNum, gpuGbColNum;
  int *gpuGbType = NULL, *gpuGbSize = NULL;

  int *gpuGbKey = NULL;
  char **gpuContent = NULL, **column = NULL;

  /*
   * @gbCount: the number of groups
   * gbConstant: whether group by constant
   */

  int gbCount;
  int gbConstant = 0;

  struct tableNode *res = (struct tableNode *)malloc(sizeof(struct tableNode));
  CHECK_POINTER(res);
  res->tupleSize = gb->tupleSize;
  res->totalAttr = gb->outputAttrNum;
  res->attrType = (int *)malloc(sizeof(int) * res->totalAttr);
  CHECK_POINTER(res->attrType);
  res->attrSize = (int *)malloc(sizeof(int) * res->totalAttr);
  CHECK_POINTER(res->attrSize);
  res->attrTotalSize = (int *)malloc(sizeof(int) * res->totalAttr);
  CHECK_POINTER(res->attrTotalSize);
  res->dataPos = (int *)malloc(sizeof(int) * res->totalAttr);
  CHECK_POINTER(res->dataPos);
  res->dataFormat = (int *)malloc(sizeof(int) * res->totalAttr);
  CHECK_POINTER(res->dataFormat);
  res->content = (char **)malloc(sizeof(char **) * res->totalAttr);
  CHECK_POINTER(res->content);

  for (int i = 0; i < res->totalAttr; i++) {
    res->attrType[i] = gb->attrType[i];
    res->attrSize[i] = gb->attrSize[i];
    res->dataFormat[i] = UNCOMPRESSED;
  }

  gpuTupleNum = gb->table->tupleNum;
  gpuGbColNum = gb->groupByColNum;

  if (gpuGbColNum == 1 && gb->groupByIndex[0] == -1) {
    gbConstant = 1;
  }

  dim3 grid(1024);
  dim3 block(128);
  int blockNum = gb->table->tupleNum / block.x + 1;
  if (blockNum < 1024)
    grid = blockNum;

  int *gpu_hashNum = NULL, *gpu_psum = NULL, *gpuGbCount = NULL;

#ifdef HAS_GMM
  CUDA_SAFE_CALL_NO_SYNC(cudaMallocEx((void **)&gpuContent, gb->table->totalAttr * sizeof(char *), FLAG_PTARRAY));
#else
  CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&gpuContent, gb->table->totalAttr * sizeof(char *)));
#endif
  column = (char **)malloc(sizeof(char *) * gb->table->totalAttr);
  CHECK_POINTER(column);

  for (int i = 0; i < gb->table->totalAttr; i++) {
    int attrSize = gb->table->attrSize[i];
    if (gb->table->dataPos[i] == MEM) {
      CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&column[i], attrSize * gb->table->tupleNum));
      CUDA_SAFE_CALL_NO_SYNC(
          hipMemcpy(column[i], gb->table->content[i], attrSize * gb->table->tupleNum, hipMemcpyHostToDevice));

      CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(&gpuContent[i], &column[i], sizeof(char *), hipMemcpyHostToDevice));
    } else {
      CUDA_SAFE_CALL_NO_SYNC(
          hipMemcpy(&gpuContent[i], &gb->table->content[i], sizeof(char *), hipMemcpyHostToDevice));
    }
  }

  if (gbConstant != 1) {

    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&gpuGbType, sizeof(int) * gb->groupByColNum));
    CUDA_SAFE_CALL_NO_SYNC(
        hipMemcpy(gpuGbType, gb->groupByType, sizeof(int) * gb->groupByColNum, hipMemcpyHostToDevice));
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&gpuGbSize, sizeof(int) * gb->groupByColNum));
    CUDA_SAFE_CALL_NO_SYNC(
        hipMemcpy(gpuGbSize, gb->groupBySize, sizeof(int) * gb->groupByColNum, hipMemcpyHostToDevice));

    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&gpuGbKey, gb->table->tupleNum * sizeof(int)));

    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&gpuGbIndex, sizeof(int) * gb->groupByColNum));
    CUDA_SAFE_CALL_NO_SYNC(
        hipMemcpy(gpuGbIndex, gb->groupByIndex, sizeof(int) * gb->groupByColNum, hipMemcpyHostToDevice));

    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&gpu_hashNum, sizeof(int) * HSIZE));
    CUDA_SAFE_CALL_NO_SYNC(hipMemset(gpu_hashNum, 0, sizeof(int) * HSIZE));

    GMM_CALL(cudaAdvise(0, CADV_INPUT | CADV_PTAINPUT));
    GMM_CALL(cudaAdvise(2, CADV_INPUT));
    GMM_CALL(cudaAdvise(3, CADV_INPUT));
    GMM_CALL(cudaAdvise(4, CADV_INPUT));
    GMM_CALL(cudaAdvise(6, CADV_OUTPUT));
    GMM_CALL(cudaAdvise(7, CADV_OUTPUT));
    build_groupby_key<<<grid, block>>>
        (gpuContent, gpuGbColNum, gpuGbIndex, gpuGbType, gpuGbSize, gpuTupleNum, gpuGbKey, gpu_hashNum);
    CUDA_SAFE_CALL_NO_SYNC(hipDeviceSynchronize());

    CUDA_SAFE_CALL_NO_SYNC(hipFree(gpuGbType));
    CUDA_SAFE_CALL_NO_SYNC(hipFree(gpuGbSize));
    CUDA_SAFE_CALL_NO_SYNC(hipFree(gpuGbIndex));

    gbCount = 1;

    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&gpuGbCount, sizeof(int)));
    CUDA_SAFE_CALL_NO_SYNC(hipMemset(gpuGbCount, 0, sizeof(int)));

    GMM_CALL(cudaAdvise(0, CADV_INPUT));
    GMM_CALL(cudaAdvise(2, CADV_DEFAULT));
    count_group_num<<<grid, block>>> (gpu_hashNum, HSIZE, gpuGbCount);
    CUDA_SAFE_CALL_NO_SYNC(hipDeviceSynchronize());

    CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(&gbCount, gpuGbCount, sizeof(int), hipMemcpyDeviceToHost));

    CUDA_SAFE_CALL(hipMalloc((void **)&gpu_psum, HSIZE * sizeof(int)));
    scanImpl(gpu_hashNum, HSIZE, gpu_psum, pp);

    CUDA_SAFE_CALL_NO_SYNC(hipFree(gpuGbCount));
    CUDA_SAFE_CALL_NO_SYNC(hipFree(gpu_hashNum));
  }

  if (gbConstant == 1)
    res->tupleNum = 1;
  else
    res->tupleNum = gbCount;

  printf("(INFO) Number of groupBy results: %d\n", res->tupleNum);

  char **gpuResult = NULL;
  char **result = NULL;

  result = (char **)malloc(sizeof(char *) * res->totalAttr);
  CHECK_POINTER(result);

#ifdef HAS_GMM
  CUDA_SAFE_CALL_NO_SYNC(cudaMallocEx((void **)&gpuResult, sizeof(char *) * res->totalAttr, FLAG_PTARRAY));
#else
  CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&gpuResult, sizeof(char *) * res->totalAttr));
#endif

  for (int i = 0; i < res->totalAttr; i++) {
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&result[i], res->tupleNum * res->attrSize[i]));
    CUDA_SAFE_CALL_NO_SYNC(hipMemset(result[i], 0, res->tupleNum * res->attrSize[i]));
    res->content[i] = result[i];
    res->dataPos[i] = GPU;
    res->attrTotalSize[i] = res->tupleNum * res->attrSize[i];
    CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(&gpuResult[i], &result[i], sizeof(char *), hipMemcpyHostToDevice));
  }

  CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&gpuGbType, sizeof(int) * res->totalAttr));
  CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(gpuGbType, res->attrType, sizeof(int) * res->totalAttr, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&gpuGbSize, sizeof(int) * res->totalAttr));
  CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(gpuGbSize, res->attrSize, sizeof(int) * res->totalAttr, hipMemcpyHostToDevice));

  struct mathExp *gpuMathExp = NULL;
  int *cpuFunc = (int *)malloc(sizeof(int) * res->totalAttr);
  int *gpuFunc = NULL;
  int *op = (int *)malloc(sizeof(int) * res->totalAttr);
  int *gpuOp = NULL;
  int *mathExpOffset = (int *)malloc(sizeof(int) * res->totalAttr);
  int *gpuMathOffset = NULL;
  int mathExpNum = 0;

  for (int i = 0; i < res->totalAttr; i++) {
    mathExpOffset[i] = mathExpNum;
    cpuFunc[i] = gb->gbExp[i].func;
    op[i] = gb->gbExp[i].exp.op;
    if (gb->gbExp[i].exp.opNum == 2)
      mathExpNum += 2;
    else
      mathExpNum += 1;
  }

  CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&gpuMathExp, sizeof(struct mathExp) * mathExpNum));
  CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&gpuMathOffset, sizeof(int) * res->totalAttr));
  CUDA_SAFE_CALL_NO_SYNC(
      hipMemcpy(gpuMathOffset, mathExpOffset, sizeof(int) * res->totalAttr, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&gpuFunc, sizeof(int) * res->totalAttr));
  CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(gpuFunc, cpuFunc, sizeof(int) * res->totalAttr, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&gpuOp, sizeof(int) * res->totalAttr));
  CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(gpuOp, op, sizeof(int) * res->totalAttr, hipMemcpyHostToDevice));

  for (int i = 0; i < res->totalAttr; i++) {
    int offset = mathExpOffset[i];
    if (gb->gbExp[i].exp.opNum == 2) {
      CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(&(gpuMathExp[offset]), (struct mathExp *)gb->gbExp[i].exp.exp,
                                        2 * sizeof(struct mathExp), hipMemcpyHostToDevice));
    } else {
      CUDA_SAFE_CALL_NO_SYNC(
          hipMemcpy(&(gpuMathExp[offset]), &(gb->gbExp[i].exp), sizeof(struct mathExp), hipMemcpyHostToDevice));
    }
  }

  free(mathExpOffset);
  free(cpuFunc);
  free(op);

  gpuGbColNum = res->totalAttr;

  if (gbConstant != 1) {
    GMM_CALL(cudaAdvise(0, CADV_INPUT | CADV_PTAINPUT));
    GMM_CALL(cudaAdvise(2, CADV_INPUT));
    GMM_CALL(cudaAdvise(3, CADV_INPUT));
    GMM_CALL(cudaAdvise(4, CADV_INPUT));
    GMM_CALL(cudaAdvise(5, CADV_INPUT));
    GMM_CALL(cudaAdvise(6, CADV_INPUT));
    GMM_CALL(cudaAdvise(7, CADV_INPUT));
    GMM_CALL(cudaAdvise(9, CADV_INPUT));
    GMM_CALL(cudaAdvise(10, CADV_INPUT));
    GMM_CALL(cudaAdvise(11, CADV_INPUT | CADV_PTADEFAULT));
    agg_cal<<<grid, block>>> (gpuContent, gpuGbColNum, gpuFunc, gpuOp, gpuMathExp, gpuMathOffset, gpuGbType,
                                gpuGbSize, gpuTupleNum, gpuGbKey, gpu_psum, gpuResult);
    CUDA_SAFE_CALL_NO_SYNC(hipFree(gpuGbKey));
    CUDA_SAFE_CALL_NO_SYNC(hipFree(gpu_psum));
  } else {
    // kaibo: gpuGbKey and gpu_psum are not allocated when gbConstant == 1, so we should not reference them in ths case
    GMM_CALL(cudaAdvise(0, CADV_INPUT | CADV_PTAINPUT));
    GMM_CALL(cudaAdvise(2, CADV_INPUT));
    GMM_CALL(cudaAdvise(3, CADV_INPUT));
    GMM_CALL(cudaAdvise(4, CADV_INPUT));
    GMM_CALL(cudaAdvise(5, CADV_INPUT));
    GMM_CALL(cudaAdvise(6, CADV_INPUT));
    GMM_CALL(cudaAdvise(7, CADV_INPUT));
    // GMM_CALL(cudaAdvise(9, CADV_INPUT));
    // GMM_CALL(cudaAdvise(10, CADV_INPUT));
    GMM_CALL(cudaAdvise(11, CADV_INPUT | CADV_PTADEFAULT));
    agg_cal_cons<<<grid, block>>> (gpuContent, gpuGbColNum, gpuFunc, gpuOp, gpuMathExp, gpuMathOffset, gpuGbType,
                                     gpuGbSize, gpuTupleNum, NULL, NULL, gpuResult);
  }

  for (int i = 0; i < gb->table->totalAttr; i++) {
    if (gb->table->dataPos[i] == MEM)
      CUDA_SAFE_CALL_NO_SYNC(hipFree(column[i]));
  }
  free(column);
  CUDA_SAFE_CALL_NO_SYNC(hipFree(gpuContent));
  CUDA_SAFE_CALL_NO_SYNC(hipFree(gpuGbType));
  CUDA_SAFE_CALL_NO_SYNC(hipFree(gpuGbSize));
  CUDA_SAFE_CALL_NO_SYNC(hipFree(gpuMathExp));
  CUDA_SAFE_CALL_NO_SYNC(hipFree(gpuMathOffset));
  CUDA_SAFE_CALL_NO_SYNC(hipFree(gpuFunc));
  CUDA_SAFE_CALL_NO_SYNC(hipFree(gpuOp));
  CUDA_SAFE_CALL_NO_SYNC(hipFree(gpuResult));

  clock_gettime(CLOCK_REALTIME, &end);
  double timeE = (end.tv_sec - start.tv_sec) * BILLION + end.tv_nsec - start.tv_nsec;
  printf("GroupBy Time: %lf\n", timeE / (1000 * 1000));

  return res;
}
