// memory region allocation
#include <stdio.h>
#include <hip/hip_runtime.h>

#include "test.h"
#include "mqx.h"

int test_malloc()
{
    size_t size_free, size_total;
    size_t size = 1024;
    void *dptr = NULL;

    if (hipMemGetInfo(&size_free, &size_total) != hipSuccess) {
        MQX_TPRINT("Cannot get device memory info");
        return -1;
    }

    while (size < size_free - 1024L * 1024L * 64L) {
        MQX_TPRINT("Allocating %lu bytes", size);
        if (hipMalloc(&dptr, size) != hipSuccess) {
            MQX_TPRINT("hipMalloc failed");
            return -1;
        }
        //mqx_print_dptr(dptr);
        if (hipFree(dptr) != hipSuccess) {
            MQX_TPRINT("hipFree failed");
            return -1;
        }
        size *= 2;
    }

    return 0;
}
