#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#ifdef USE_MQX
#include "mqx.h"
#endif

#define TVAL(t)         ((t).tv_sec * 1000.0 + (t).tv_usec / 1000.0)
#define TDIFF(t1, t2)   (TVAL(t2) - TVAL(t1))

#ifndef CUDA_SAFE_CALL
#define CUDA_SAFE_CALL(call) \
    do { \
        hipError_t err = call; \
        if(hipSuccess != err) { \
            fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n", \
                    __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(EXIT_FAILURE); \
        } \
    } while (0)
#endif

#define BLOCK_SIZE  16
#define WIDTH       (BLOCK_SIZE * 128)
#define HEIGHT      WIDTH

// Allocates a matrix with random float entries.
void randomInit(float* data, int size)
{
    for (int i = 0; i < size; ++i)
        data[i] = rand() / (float)RAND_MAX;
}

//float multiplication kernel called by MatMul()
__global__ void MatMulKernel(float *A, float *B, float *C)
{
	// Each thread computes one element of C by accumulating results into Cvalue
	float Cvalue = 0;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	for (int e = 0; e < WIDTH; ++e)
		Cvalue += A[row * WIDTH + e] * B[e * WIDTH + col];
	C[row * WIDTH + col] = Cvalue;
}

// float multiplication - Host code
// float dimensions are assumed to be multiples of BLOCK_SIZE
void MatMul(const float *A, const float *B, float *C)
{
	size_t size = WIDTH * HEIGHT * sizeof(float);
	float *d_A, *d_B, *d_C;
	struct timeval t1, t2;

	gettimeofday(&t1, NULL);

	// Load A and B to device memory
	CUDA_SAFE_CALL(hipMalloc((void**)&d_A, size));
	CUDA_SAFE_CALL(hipMemcpy(d_A, A, size, hipMemcpyHostToDevice));

	CUDA_SAFE_CALL(hipMalloc((void**)&d_B, size));
	CUDA_SAFE_CALL(hipMemcpy(d_B, B, size, hipMemcpyHostToDevice));

	// Allocate C in device memory
	CUDA_SAFE_CALL(hipMalloc((void**)&d_C, size));

	// Invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(WIDTH / dimBlock.x, HEIGHT / dimBlock.y);

#ifdef USE_MQX
	CUDA_SAFE_CALL(cudaAdvise(0, CADV_INPUT));
	CUDA_SAFE_CALL(cudaAdvise(1, CADV_INPUT));
	CUDA_SAFE_CALL(cudaAdvise(2, CADV_OUTPUT));
#endif
	MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
	CUDA_SAFE_CALL(hipDeviceSynchronize());

	// Read C from device memory
	hipMemcpy(C, d_C, size,hipMemcpyDeviceToHost);

	// Free device memory
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	gettimeofday(&t2, NULL);
	printf("Matrix multiplication took %f ms\n", TDIFF(t1, t2));
}

int main(int argc, char* argv[])
{
	unsigned int size = WIDTH*HEIGHT;
	unsigned int mem_size = sizeof(float) * size;
	float *h_A, *h_B, *h_C;

	// Allocate host memory for matrices A and B
	h_A = (float*)malloc(mem_size);
	h_B = (float*)malloc(mem_size);
	h_C = (float*)malloc(mem_size);

	// set seed for rand()
	srand(2014);

	// initialize host memory
	randomInit(h_A, size);
	randomInit(h_B, size);

	//invoke MatMul
	MatMul(h_A,h_B,h_C);

	free(h_C);
	free(h_B);
	free(h_A);
	return 0;
}
